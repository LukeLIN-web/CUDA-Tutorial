//learn how to write a simple CUDA program that performs matrix multiplication.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define TILE_WIDTH 16

__global__ void matrixMul(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

void matrixMulCPU(float *A, float *B, float *C, int m, int n, int k) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            float sum = 0.0f;
            for (int p = 0; p < n; ++p) {
                sum += A[i * n + p] * B[p * k + j];
            }
            C[i * k + j] = sum;
        }
    }
}

int main() {
    int m = 512;
    int n = 512;
    int k = 512;

    float *h_A, *h_B, *h_C_CPU, *h_C_CUDA;

    size_t size_A = m * n * sizeof(float);
    size_t size_B = n * k * sizeof(float);
    size_t size_C = m * k * sizeof(float);


    hipEvent_t startWMMA;
    hipEvent_t stopWMMA;
    
    hipEventCreate(&startWMMA);
    hipEventCreate(&stopWMMA);

    // Allocate host memory
    h_A = (float *)malloc(size_A);
    h_B = (float *)malloc(size_B);
    h_C_CPU = (float *)malloc(size_C);
    h_C_CUDA = (float *)malloc(size_C);

    // Initialize matrices A and B
    for (int i = 0; i < m * n; ++i) {
        h_A[i] = 1.0f;
    }
    for (int i = 0; i < n * k; ++i) {
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    // Transfer data from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((k + TILE_WIDTH - 1) / TILE_WIDTH, (m + TILE_WIDTH - 1) / TILE_WIDTH);

    // Launch kernel
    hipEventRecord(startWMMA);
    matrixMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);
    hipEventRecord(stopWMMA);
    hipEventSynchronize(stopWMMA);
    
    // Transfer results from device to host
    hipMemcpy(h_C_CUDA, d_C, size_C, hipMemcpyDeviceToHost);

    struct timeval start;
    struct timeval end;
    gettimeofday(&start, NULL);
    matrixMulCPU(h_A, h_B, h_C_CPU, m, n, k);
    gettimeofday(&end, NULL);
    long long elapsed = (end.tv_sec - start.tv_sec) * 1000000LL + (end.tv_usec - start.tv_usec);
    

    // Compare results from CPU and CUDA
    bool isEqual = true;
    for (int i = 0; i < m * k; ++i) {
        if (h_C_CPU[i] != h_C_CUDA[i]) {
            isEqual = false;
            break;
        }
    }

    if (isEqual) {
        printf("Results match between CPU and CUDA.\n");
        float gpuTime;
        hipEventElapsedTime(&gpuTime, startWMMA, stopWMMA);
        printf("GPU took %fms\n", gpuTime);
        printf("CPU elapsed time = %lld ms\n", elapsed/1000);
    } else {
        printf("Results do not match between CPU and CUDA.\n");
        
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C_CPU);
    free(h_C_CUDA);

    return 0;
}
